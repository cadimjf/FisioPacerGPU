#include "hip/hip_runtime.h"
﻿#include "kernel.h"
#include <vector>


inline __device__ __host__ int testedef(){return 0;}

#define chker(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


typ_ca *deviceCA;
typ_stats *deviceStats;
typ_press *devicePressureCA;
typ_param *deviceParams;
typ_point *devicePntsNew;//
typ_point *devicePntsOld;//
typ_point *devicePntsIntrm;//

void deviceAlloc(typ_ca *CA) {

    chker(hipMalloc((void**)&deviceCA,         sizeof(typ_ca)));
    chker(hipMalloc((void**)&deviceStats,      sizeof(typ_stats)));
    chker(hipMalloc((void**)&devicePressureCA, sizeof(typ_press)));
    chker(hipMalloc((void**)&deviceParams,     sizeof(typ_param)));
    chker(hipMalloc((void**)&devicePntsNew,    sizeof(typ_point) * CA->params->pointsNum));
    chker(hipMalloc((void**)&devicePntsOld,    sizeof(typ_point) * CA->params->pointsNum));
    chker(hipMalloc((void**)&devicePntsIntrm,  sizeof(typ_point) * CA->params->pointsNum));

    /*
    typ_face** aFaces;

    int stimSize;
    t_stim** aStim; //
    int nRegions;
    t_par_ac** aParam;//
    //pressure faces
    int numFaces;
    typ_face** aFaces;//*/



    //gpuErrchk(hipMalloc((void**)&CA_dev->params, sizeof(typ_param));
    //CA->params->aParam = (t_par_ac**)malloc(sizeof(t_par_ac*) * CA->params->nRegions);

    //gpuErrchk(hipMalloc((void**)&CA_dev->params->aParam));
}

void deviceCopy(typ_ca *hCA) {
    chker(hipMemcpy(deviceCA,         hCA,                 sizeof(typ_ca),     hipMemcpyHostToDevice));
    chker(hipMemcpy(deviceStats,      hCA->stats,          sizeof(typ_stats),  hipMemcpyHostToDevice));
    chker(hipMemcpy(devicePressureCA, hCA->pressureCA,     sizeof(typ_press),  hipMemcpyHostToDevice));
    chker(hipMemcpy(deviceParams,     hCA->params,         sizeof(typ_param),  hipMemcpyHostToDevice));
    int pn = hCA->params->pointsNum;
    chker(hipMemcpy(devicePntsNew,    hCA->pnts_new,       sizeof(typ_point) * pn, hipMemcpyHostToDevice));
    chker(hipMemcpy(devicePntsOld,    hCA->pnts_old,       sizeof(typ_point) * pn, hipMemcpyHostToDevice));
    chker(hipMemcpy(devicePntsIntrm,  hCA->pnts_intrm,     sizeof(typ_point) * pn, hipMemcpyHostToDevice));
    
   
}


__global__ void teste(
    typ_ca* dCA,
    typ_stats* dStats,
    typ_press* dPressureCA,
    typ_point* dPntsNew, typ_point* dPntsOld, typ_point* devPntsIntrm)
{
    testedef();
    //int i = threadIdx.x;
    //devCA->time = deviceAFaces[0]->pt1;
}


void deviceDealloc( ) {
    chker(hipFree(deviceStats));
    chker(hipFree(devicePressureCA));
    chker(hipFree(deviceCA));
    chker(hipFree(deviceParams));

    chker(hipFree(devicePntsIntrm));
    chker(hipFree(devicePntsOld));
    chker(hipFree(devicePntsNew));
}

/*
typedef struct str_cellularautomata{
    typ_t0_element   **ini;//
    typ_dt_element **t_old;//
    typ_dt_element **t_new;//
    lst_item **omega_b;  //
    lst_item **omega_a;//
    typ_param* params;//
    typ_point **pnts_new;//
    typ_point **pnts_old;//
    typ_point **pnts_intrm;//
}typ_ca;// automato cellular

*/


/**
 *
 * @param CA
 */
void initializeCA(typ_ca* CA)
{
    for (int i = 0; i < CA->params->elementsNum; i++)
    {
        iniVolumes(i, CA);
    }
    double mass = 0.0;
    //iterate over points to find their masses
    for (int i = 0; i < CA->params->pointsNum; i++) {
        mass = 0.0;
        lst_item* cur = CA->omega_b[i];
        while (cur != NULL) {
            mass += CA->ini[cur->value]->volCel_ini;
            cur = cur->next;
        }
        CA->pnts_new[i].mass = CA->pnts_old[i].mass = ro_mass_dens * mass / 4.0;
    }
    for (int i = 0; i < CA->params->elementsNum; i++)
    {
        CA->t_old[i]->cellT = CA->t_new[i]->cellT = 0.0f;
        CA->t_old[i]->ECTNC_force_t_ini = CA->t_new[i]->ECTNC_force_t_ini = 0.0f;
        CA->t_old[i]->ECTNC_force_val_ini = CA->t_new[i]->ECTNC_force_val_ini = 0.0f;
        CA->t_old[i]->ECTNC_ap_t_ini = CA->t_new[i]->ECTNC_ap_t_ini = 0.0f;
        CA->t_old[i]->ECTNC_ap_val_ini = CA->t_new[i]->ECTNC_ap_val_ini = 0.0f;
        CA->t_old[i]->ECTNC_ap_t_end = CA->t_new[i]->ECTNC_ap_t_end = 0.0f;
        CA->t_old[i]->ECTNC_force_t_end = CA->t_new[i]->ECTNC_force_t_end = 0.0f;

        //initially, all cells are healthy
        CA->ini[i]->cellCond = HEALTHY;
        //V state
        CA->t_new[i]->V_state = CA->t_old[i]->V_state = V0;
        //F state
        CA->t_new[i]->F_state = CA->t_old[i]->F_state = F0;
        iniGeometry(i, CA);
        //sets the pacemaker up
        for (int iS = 0; iS < CA->params->stimSize; iS++) {
            t_stim* s = CA->params->aStim[iS];
            if (
                CA->t_old[i]->bary[0] > s->iniX && CA->t_old[i]->bary[0] < s->endX &&
                CA->t_old[i]->bary[1] > s->iniY && CA->t_old[i]->bary[1] < s->endY &&
                CA->t_old[i]->bary[2] > s->iniZ && CA->t_old[i]->bary[2] < s->endZ
                ) {
                CA->ini[i]->cellCond = paceMaker;
                CA->ini[i]->pmRegion = iS;
            }
        }
        restartAPDElectroTonic(i, CA);
        CA->t_old[i]->APTime1 = CA->t_new[i]->APTime1;
        CA->t_old[i]->APTime2 = CA->t_new[i]->APTime2;
        CA->t_old[i]->APTime3 = CA->t_new[i]->APTime3;
        CA->t_old[i]->APTime4 = CA->t_new[i]->APTime4;

    }
    iniPressure(CA);
}


/**
 *
 * @param CA
 * @param strFolderOut
 * @param finalTime
 * @param save
 * @param numThreads
 * @param threadsByIndividual
 * @return
 */
int simulate(typ_ca* CA, bool save) {
    //time units in SECONDS
    CA->time = 0.0;
    CA->timeSaving = 0.0;
    CA->stats->contSave = 0;

    CA->stats->minVol = DBL_MAX;
    CA->stats->maxVol = 0.0;

    int count = 0;
    typ_point* aux3;
    typ_dt_element** auxCA;
    char filename[255];
    sprintf(filename, "%sfisiopacer.txt", CA->params->outputFolder/*.c_str()*/);
    FILE* fileDt = fopen(filename, "w+");

    double* forcesOnPts_interm = (double*)malloc(sizeof(double) * CA->params->pointsNum * 3);
    if (forcesOnPts_interm == NULL) {
        throw MyException("Allocation failure for forcesOnPts_interm.", __FILE__, __LINE__);
    }
    double* forcesOnPts = (double*)malloc(sizeof(double) * CA->params->pointsNum * 3);
    if (forcesOnPts == NULL) {
        throw MyException("Allocation failure for forcesOnPts.", __FILE__, __LINE__);
    }
    for (int k = 0; k < CA->params->pointsNum; k++) {
        forcesOnPts[I2d(k, 0, 3)] = forcesOnPts[I2d(k, 1, 3)] = forcesOnPts[I2d(k, 2, 3)] = 0.0f;
        forcesOnPts_interm[I2d(k, 0, 3)] = forcesOnPts_interm[I2d(k, 1, 3)] = forcesOnPts_interm[I2d(k, 2, 3)] = 0.0f;
    }
    initializeCA(CA);
    double sumDt = 0.0;
    CA->stats->volIni = 0.0;
    for (int i = 0; i < CA->params->elementsNum; i++) {
        CA->stats->volIni += CA->t_old[i]->volCel;
    }
    CA->volume = CA->stats->volIni;
    int retValFinal = 0;

    CA->stats->maxDeltaVol = 0.0;
    CA->stats->volMaxDelta = 0.0;
    CA->stats->avgVel = 0.0;


    stats(CA, forcesOnPts);

    deviceAlloc(CA);
    deviceCopy(CA);

    cout << "vol ini antes: "<<CA->stats->volIni << endl;
    teste <<< 1, 1>>> (deviceCA, deviceStats, devicePressureCA, devicePntsNew, devicePntsOld, devicePntsIntrm);
    hipMemcpy(CA->stats, deviceStats, sizeof(typ_stats), hipMemcpyDeviceToHost);
    hipMemcpy(CA, deviceCA, sizeof(typ_ca), hipMemcpyDeviceToHost);
    
    hipMemcpy(CA->pnts_intrm, devicePntsIntrm, sizeof(typ_point)*CA->params->pointsNum, hipMemcpyDeviceToHost);
    
    cout << "time: "<<CA->time << endl;
    cout << "volini depois : " << CA->stats->volIni << endl;
    deviceDealloc();

    exit(0);

    //cout<<"VOlume ini " << CA->volume<<endl;
    while (CA->time <= CA->params->simulationTime)
    {
        simulationStep(CA, forcesOnPts);
        if (CA->params->mecSim == 1) {
            // EulerMethod(CA, forcesOnPts);
            VelocityVerletMethod(CA, forcesOnPts, forcesOnPts_interm);
        }
        incPressureStates(CA);
        if (save) {
            save_step(fileDt, CA, CA->params->outputFolder, forcesOnPts);
        }
        for (int k = 0; k < CA->params->pointsNum; k++) {
            forcesOnPts[I2d(k, 0, 3)] = forcesOnPts[I2d(k, 1, 3)] = forcesOnPts[I2d(k, 2, 3)] = 0.0f;
        }
        CA->time += CA->params->dt;
        auxCA = CA->t_old;
        CA->t_old = CA->t_new;
        CA->t_new = auxCA;
        //
        aux3 = CA->pnts_old;
        CA->pnts_old = CA->pnts_new;
        CA->pnts_new = aux3;
        //
        count++;
        sumDt += CA->params->dt;
    }// fim do while
    fclose(fileDt);
    //saveAPD(CA, strFolderOut, filename);
    if (forcesOnPts != NULL)
        free(forcesOnPts);
    forcesOnPts = NULL;
    if (forcesOnPts_interm != NULL)
        free(forcesOnPts_interm);
    forcesOnPts_interm = NULL;
    if (CA->params->printOutput == 1) {
        printf("final time: %.3f\n", CA->time);
        printf("Volume: [%.3f %.3f] [%.3e %.3e] \n", CA->stats->minVol / CA->stats->volIni * 100.0, CA->stats->maxVol / CA->stats->volIni * 100.0, CA->stats->volIni, CA->volume);
        printf("iterações: %d. Dt medio %g\n", count, sumDt / count);
    }

    return retValFinal;
}

/**
 *
 * @param nThreads
 * @param POINTS_OLD
 * @param forcesOnPts
 * @param time
 * @return
 */
void simulationStep(typ_ca* CA,
    double* forcesOnPts)
{
    double volT = 0.0;
    int contCA = 0;
    //#pragma omp parallel for schedule(static) num_threads(nThreads) reduction(+:volT)
    for (int i = 0; i < CA->params->elementsNum; i++) {
        if (CA->params->paSim == 1) {
            CAStep_i(i, CA);
            contCA++;
            computeNewAPDElectroTonic(i, CA);
        }
        computeForceOnElement(CA, forcesOnPts, i);
        //verifica se o volume da celula é menor que 1% do inicial - netste caso, mata o processo pq é sinal de erro.
        if (CA->t_new[i]->volCel < 0.001 * CA->ini[i]->volCel_ini) {
            CA->params->mecSim = 0;
            CA->params->paSim = 0;
            if (CA->params->printOutput == 1) {
                cout << "Mata por volume pequeno[" << i << "]: " << CA->time << endl;
                cout << "Inicial: " << CA->ini[i]->volCel_ini << " | Atual: " << CA->t_new[i]->volCel << endl;
                cout << CA->pnts_old[CA->ini[i]->iPt1].x << " " <<
                    CA->pnts_old[CA->ini[i]->iPt1].y << " " <<
                    CA->pnts_old[CA->ini[i]->iPt1].z << endl;
                cout << CA->pnts_old[CA->ini[i]->iPt2].x << " " <<
                    CA->pnts_old[CA->ini[i]->iPt2].y << " " <<
                    CA->pnts_old[CA->ini[i]->iPt2].z << endl;

                throw MyException("1percent volume.", __FILE__, __LINE__);

            }
        }
        volT += CA->t_new[i]->volCel;
    }//fim pragma
    //exit(0);
    if (CA->params->mecSim == 1) {
        computePressurePoints(CA, forcesOnPts);
    }
    CA->volume = volT;
}


/**
 *
 * @param CA
 * @param forcesOnPts
 */
void stats(typ_ca* CA, double* forcesOnPts) {
    if (CA->volume > CA->stats->maxVol) CA->stats->maxVol = CA->volume;
    if (CA->volume < CA->stats->minVol) CA->stats->minVol = CA->volume;
    double deltaVol = fabs(CA->volume - CA->stats->volIni);
    if (deltaVol > CA->stats->maxDeltaVol) {
        CA->stats->volMaxDelta = CA->volume;
        CA->stats->maxDeltaVol = deltaVol;
    }
    CA->stats->max[0] = CA->stats->max[1] = CA->stats->max[2] = -DBL_MAX;
    CA->stats->min[0] = CA->stats->min[1] = CA->stats->min[2] = +DBL_MAX;
    double sumVel = 0.0;
    for (int k = 0; k < CA->params->pointsNum; k++) {
        //Zero the forces
        //Essa linha foi pra o método de verlet forcesOnPts[I2d(k,0,3)]=forcesOnPts[I2d(k,1,3)]=forcesOnPts[I2d(k,2,3)]=0.0f;
        //
        CA->stats->min[0] = min(CA->stats->min[0], CA->pnts_old[k].x);
        CA->stats->min[1] = min(CA->stats->min[1], CA->pnts_old[k].y);
        CA->stats->min[2] = min(CA->stats->min[2], CA->pnts_old[k].z);
        CA->stats->max[0] = my_max(CA->stats->max[0], CA->pnts_old[k].x);
        CA->stats->max[1] = my_max(CA->stats->max[1], CA->pnts_old[k].y);
        CA->stats->max[2] = my_max(CA->stats->max[2], CA->pnts_old[k].z);

        double v[3] = { CA->pnts_old[k].xV, CA->pnts_old[k].yV, CA->pnts_old[k].zV };
        sumVel += my_norm(v);
    }
    CA->stats->avgVel = sumVel / CA->params->pointsNum;
}



/**
 *
 * @param numThreads
 * @param simulationTime
 * @param inputCarpFolder
 * @param outputFolder
 * @param paramFile
 * @param save
 * @param numSolutions
 * @param numRefPts
 * @param aRefIDPts
 * @param timeSolutions
 * @param aRefPts
 * @param threadsByIndividual
 * @return
 */
int startCA(string paramFile, bool save)
{
    try {


        //
        typ_ca* CA = (typ_ca*)malloc(sizeof(typ_ca));
        if (CA == NULL) {
            throw MyException("Allocation failure for CA.", __FILE__, __LINE__);
        }
        CA->params = (typ_param*)malloc(sizeof(typ_param));
        if (CA->params == NULL) {
            throw MyException("Allocation failure for parameter structure.", __FILE__, __LINE__);
        }
        readParameterFile(paramFile, CA);
        allocCA(CA);
        int retSim = simulate(CA, save);

        deallocCA(CA);
        return retSim;
    }
    catch (MyException& caught) {
        std::cout << caught.getMessage() << std::endl;
    }
    return 0;
}
/**
 *
 * @param CA
 */
void deallocCA(typ_ca* CA) {
    try {
        myArrayDeallocation<typ_dt_element>(CA->t_old, CA->params->elementsNum);
        myArrayDeallocation<typ_dt_element>(CA->t_new, CA->params->elementsNum);
        myArrayDeallocation<typ_t0_element>(CA->ini, CA->params->elementsNum);
        //
        if (CA->pnts_new    != NULL) free(CA->pnts_new);
        if (CA->pnts_old    != NULL) free(CA->pnts_old);
        if (CA->pnts_intrm  != NULL) free(CA->pnts_intrm);
        //
        freeAList(CA->omega_a, CA->params->elementsNum);
        freeAList(CA->omega_b, CA->params->pointsNum);

        for (int i = 0; i < CA->params->stimSize; i++) {
            if (CA->params->aStim[i] != NULL) free(CA->params->aStim[i]);
        }
        if (CA->params->aStim != NULL) free(CA->params->aStim);
        //
        for (int i = 0; i < CA->params->numFaces; i++) {
            if (CA->params->aFaces[i] != NULL) free(CA->params->aFaces[i]);
        }
        if (CA->params->aFaces != NULL) free(CA->params->aFaces);
        //

        for (int i = 0; i < CA->params->nRegions; i++) {
            if (CA->params->aParam != NULL) {
                if (CA->params->aParam[i] != NULL) {
                    free(CA->params->aParam[i]);
                }
            }
        }
        if (CA->params->aParam != NULL) free(CA->params->aParam);
        if (CA->params != NULL) free(CA->params);
        if (CA->stats != NULL) free(CA->stats);
        if (CA->pressureCA != NULL) free(CA->pressureCA);
        if (CA != NULL) free(CA);
    }
    catch (MyException& caught) {
        std::cout << caught.getMessage() << std::endl;
    }
}
/**
 *
 * @param strPtsFile
 * @param strEleFile
 * @param paramFile
 * @param strStmFile
 * @param strFibFile
 * @param strBoundFile
 * @param strPressFile
 * @return
 */
void allocCA(typ_ca* CA) {
    //input data
    string strPtsFile = CA->params->inputFolder;
    strPtsFile.append(".pts");
    //
    string strEleFile = CA->params->inputFolder;
    strEleFile.append(".elem");
    //
    string strFibFile = CA->params->inputFolder;
    strFibFile.append(".fib");
    //
    string strStmFile = CA->params->inputFolder;
    strStmFile.append(".stim");
    //
    string strBoundFile = CA->params->inputFolder;
    strBoundFile.append(".bound");
    //
    string strPressFile = CA->params->inputFolder;
    strPressFile.append(".press");

    try {
        //le o tamanho das malhas nos arquivos de entrada
        CA->params->pointsNum = readSize(strPtsFile);
        CA->params->elementsNum = readSize(strEleFile);
        
        
        CA->omega_a = iniAList(CA->params->elementsNum);
        if (CA->omega_a == NULL) {
            throw MyException("Allocation failure for CA->omega_a.", __FILE__, __LINE__);
        }
        CA->omega_b = iniAList(CA->params->pointsNum);
        if (CA->omega_b == NULL) {
            throw MyException("Allocation failure for CA->omega_b.", __FILE__, __LINE__);
        }
        //points
        CA->pnts_new = (typ_point*)malloc(CA->params->pointsNum * sizeof(typ_point));
        if (CA->pnts_new == NULL) {
            throw MyException("Allocation failure for CA->pnts_new.", __FILE__, __LINE__);
        }
        CA->pnts_old = (typ_point*)malloc(CA->params->pointsNum * sizeof(typ_point));
        if (CA->pnts_old == NULL) {
            throw MyException("Allocation failure for CA->pnts_old.", __FILE__, __LINE__);
        }
        CA->pnts_intrm = (typ_point*)malloc(CA->params->pointsNum * sizeof(typ_point));
        if (CA->pnts_intrm == NULL) {
            throw MyException("Allocation failure for CA->pnts_intrm.", __FILE__, __LINE__);
        }
        //elements
        CA->t_old = myArrayAllocation<typ_dt_element>(CA->params->elementsNum);
        if (CA->t_old == NULL) {
            throw MyException("Allocation failure for CA->t_old.", __FILE__, __LINE__);
        }
        CA->t_new = myArrayAllocation<typ_dt_element>(CA->params->elementsNum);
        if (CA->t_new == NULL) {
            throw MyException("Allocation failure for CA->t_new.", __FILE__, __LINE__);
        }
        CA->ini = myArrayAllocation<typ_t0_element>(CA->params->elementsNum);
        if (CA->ini == NULL) {
            throw MyException("Allocation failure for CA->ini.", __FILE__, __LINE__);
        }
        //

        CA->stats = (typ_stats*)malloc(sizeof(typ_stats));
        if (CA->stats == NULL) {
            throw MyException("Allocation failure for stats structure.", __FILE__, __LINE__);
        }
        //opens the files and fill arrays
        openFile(CA, strPtsFile, strEleFile, strFibFile, strBoundFile, strPressFile, strStmFile);
        CA->pressureCA = (typ_press*)malloc(sizeof(typ_press));
        if (CA->pressureCA == NULL) {
            throw MyException("Allocation failure for stats structure.", __FILE__, __LINE__);
        }
    }
    catch (MyException& caught) {
        std::cout << caught.getMessage() << std::endl;
    }
}

/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/
